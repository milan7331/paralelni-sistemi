#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "januar2023_zadatak1.cuh"

// Neka je dat CUDA kernel i odgovarajuća funkcija koja ga poziva.
// __global__ void vecAddKernel(float* A_d, float* B_d, float* C_d, int n)
// {
//     int i = threadIdx.x + blockDim.x * blockIdx.x;
//     if (i < n)
//     {
//         C_d[i] = A_d[i] + B_d[i];
//     }
// }
//   
// int vecAdd(float* A, float* B, float* C, int n) //pordrazumevati da su nizovi A,B i C dužine n
// {
//     int size = n * sizeof(float);
//     hipMalloc((void**)&A_d, size);
//     hipMalloc((void**)&B_d, size);
//     hipMalloc((void**)&C_d, size);
//     hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
//     hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
//     vecAddKernel << <ceil(n / 256), 256 >> > (A_d, B_d, C_d, n);
//     hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
// }
// 
// a.  Ako su A, B, C nizovi od 1000 elemenata, koliko će blokova niti biti pokrenuto?
// b.  Ako su A, B, C nizovi od 1000 elemenata, koliko će warp-ova biti u svakom od blokova?
// c.  Ako su A, B, C nizovi od 1000 elemenata, koliko će niti biti u gridu?
// d.  Ako su A, B, C nizovi od 1000 elemenata, da li će postojati divergentnost u izvršenju kernela? Ako da, u kojoj
//     liniji. Objasniti zašto.
// e.  Ako su A, B, C nizovi od 768 elemenata, da li će postojati divergentnost u izvršenju kernela? Ako da, u kojoj
//     liniji. Objasniti zašto.
// f.  Šta dati kernel radi?
// g.  Modifikovati kernel tako da svaki 256. element rezultujućeg niza sadrži srednju vrednost prethodnih 255
//     elemenata.
//
//     Napomena: kod će biti izmenjen u odnosu na originalni tako da je moguće pokrenuti aplikaciju.
//     Rešenje:
// 
// a.  Biće pokrenuto 4 bloka.
// b.  Ako uzmemo da blok ima max veličinu od 256 tredova, i da warp standardno čine 32 niti, dolazimo do zaključka da
//     jedan blok ima 8 warpova.
// c.  Broj niti u gridu je 256 * 4 = 1024
// d.  Da, do divergentnosti dolazi u liniji koja ima naredbu grananja. Konkretno u četvrtom bloku imamo 24 niti koje
//     neće raditi ništa, samo će čekati da ostale niti završe svoje operacije.
// e.  Ne, do divergentnosti može doći u liniji koja ima naredbu granaja. U našem slučaju sve niti će biti upošljene.
// f.  Svaka nit koja izvršava kernel računa index na osnovu koga pristupa nizovima. Indeksi su u odnosu na dimenzije
//     grida i bloka raspoređeni "linearno". Svaka nit je zadužena za samo jedan indeks, dva učitavanja i jedna operacija
//     sabiranja. Ove operacije se izvršavaju samo ako je indeks niti manji od veličine niza. Ostale niti ne rade ništa.
// g.  Rešenje je priloženo ispod.
// 

#define BLK_SIZE 256                    // broj tredova u jednom bloku

__global__ void vecAddKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        C_d[i] = A_d[i] + B_d[i];       // koristi se globalna memorija koja je spora
    }
    
}

__global__ void vecAddKernelModified(float* A_d, float* B_d, float* C_d, int n)
{
    __shared__ float C_temp[BLK_SIZE];  // koristimo deljivu memoriju na nivou bloka

    // inicijalizacija lokalne promenljive koja ima realnu lokaciju niti u odnosu na grid, korisno zbog pristupa nizu
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // inicijalno učitavanje podataka u privremenu deljenu memoriju
    if (index < n)
    {
        C_temp[threadIdx.x] = A_d[index] + B_d[index];
    }
    __syncthreads();

    // poslednja nit iz bloka računa srednju vrednost prethodnih niti. Ovo je jedini razlog zašto koristimo deljivu
    // memoriju, na ovaj način izbegavamo ponovno obraćanje globalnoj memoriji koja je znatno sporija.
    if (index < n && threadIdx.x + 1 == blockDim.x)
    {
        for (int i = 0; i < threadIdx.x; i++)
        {
            C_temp[threadIdx.x] += C_temp[i];
        }

        C_temp[threadIdx.x] /= 255;
    }
    __syncthreads();
    
    // na kraju upisujemo izmenjeni niz u globalnu memoriju
    if (index < n)
    {
        C_d[index] = C_temp[threadIdx.x];
    }
}

static int vecAdd(float* A, float* B, float* C, int n)
{
    // Napomena: funkcija je izmenjena da bi mogla da se pokrene ispravno.

    float* A_d = nullptr;               // pokazivač na niz A u memoriji device-a / grafičke kartice
    float* B_d = nullptr;               // pokazivač na niz B u memoriji device-a / grafičke kartice
    float* C_d = nullptr;               // pokazivač na niz C u memoriji device-a / grafičke kartice

    int size;                           // veličina niza u bajtovima
    int blockCount;                     // potreban broj blokova se računa u odnosu na broj elemenata zadatih nizova
                                        // zbog ograničenja broja tredova po bloku
        
    // inicijalizacija
    size = n * sizeof(float);
    blockCount = (int)ceil((float)n / (float)BLK_SIZE);

    // cuda alokacija memorije i prenos nizova u globalnu memoriju grafičke kartice
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    // poziv samog kernela koji se izvršava na kartici
    vecAddKernelModified<<<blockCount, BLK_SIZE>>>(A_d, B_d, C_d, n);

    // kopiranje rezultata nazad u glavnu memoriju procesora
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    // oslobađanje memorije i reset grafičke kartice
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipDeviceReset();

    return 0;
}

static void TestResults(float* A, float* B, float* C, int n)
{
    // Napomena: skroz nepotrebna funkcija, služi samo za proveru tačnosti prilikom pisanja koda

    unsigned int error_count = 0;       // broj detektovanih grešaka 

    // iteracija kroz nizove kako bi utvrdili tačnost
    for (auto i = 0; i < n; i++)
    {
        if (A[i] + B[i] != C[i] && (i+1) % 256 != 0)
        {
            error_count++;
        }
    }

    // ispis rezultata u konzoli
    std::cout << "Total error count: " << error_count << std::endl;

    std::cout << "The first 10 numbers from each array: " << std::endl;
    for (int i = 0; i < 10; i++)
    {
        std::cout << i << " : " << A[i] << " " << B[i] << " " << C[i] << "\n";
    }

    std::cout << "Element 256 = " << C[255] << "\n";
    std::cout << "Element 512 = " << C[511] << "\n";
    std::cout << "Element 768 = " << C[767] << "\n";
}

int januar2023_zadatak1()
{
    const int n = 1000;                 // broj elemenata u svakom nizu
    float A_h[n]{};                     // niz A u glavnoj memoriji procesora ("hosta")
    float B_h[n]{};                     // niz B u glavnoj memoriji procesora ("hosta")
    float C_h[n]{};                     // niz C u glavnoj memoriji procesora ("hosta")

    // inicijalizacija nizova 
    for (auto i = 0; i < n; A_h[i++] = (float)(rand() % 10));
    for (auto i = 0; i < n; B_h[i++] = i);

    // poziv wrapper funkcije koja priprema i poziva kernel funkciju
    vecAdd(A_h, B_h, C_h, n);

    // dodatna metoda za proveru tačnosti - nije potrebna
    TestResults(A_h, B_h, C_h, n);

    return 0;
}
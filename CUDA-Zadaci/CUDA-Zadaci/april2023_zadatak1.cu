#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "april2023_zadatak1.cuh"

// Anti-Aliasing je tehnika kojom se popravlja kvalitet slika niske rezolucije. Postoje različiti algoritmi koji
// rešavaju ovaj problem u zavisnosti od situacije. Neka je dat specifičan slučaj gde je, kao matrica nula i jedinica,
// data uvećana crno-bela slika koja predstavlja potpuno crno slovo (1) na potpuno belom papiru (0). Nad ovom slikom
// potrebno je primeniti jednostavnu Anti-Aliasing tehniku, koja preračunava nove boje svakog piksela kao srednju
// vrednost njegove boje i boje svih 8 susednih piksela (ako ne postoji 8 susednih, uzima se onoliko koliko ih ima).
// Proceduru treba ponoviti k puta. Koristeći CUDA tehnologiju, napisati paralelni program koji što efikasnije izvršava
// ovu transformaciju.

#define PICTURE_DIM 128
#define BLOCK_DIM 32 // ne preko 32 zbog deljene memorije, može da se radi veći blok ali bez deljene

__global__ void AntiAliasingKernel(float* pic_device_in, float* pic_device_out, int k)
{
	__shared__ float  pic_shared_step1[(BLOCK_DIM + 2) * (BLOCK_DIM + 2)];
	__shared__ float  pic_shared_step2[(BLOCK_DIM + 2) * (BLOCK_DIM + 2)];
	
	// Calculate global indices
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;

	// Calculate shared memory indices with padding
	unsigned int shared_index_x = threadIdx.x + 1;
	unsigned int shared_index_y = threadIdx.y + 1;

	// Calculate global index for current thread
	unsigned int index = idy * blockDim.x * gridDim.x + idx;

	// Load pixel data into shared memory
	pic_shared_step1[shared_index_y * (BLOCK_DIM + 2) + shared_index_x] = pic_device_in[index];

	// Edge loading - handle edges by loading additional pixels
	if (threadIdx.x == 0)
	{
		pic_shared_step1[shared_index_y * (BLOCK_DIM + 2)] = pic_device_in[index - 1]; // load left edge
	}
	else if (threadIdx.x == blockDim.x - 1)
	{
		pic_shared_step1[shared_index_y * (BLOCK_DIM + 2) + blockDim.x + 1] = pic_device_in[index + 1]; // load right edge
	}

	// Load top and bottom edges
	if (threadIdx.y == 0)
	{
		pic_shared_step1[shared_index_x] = pic_device_in[index - blockDim.x]; // load top edge
	}
	else if (threadIdx.y == blockDim.y - 1)
	{
		pic_shared_step1[(BLOCK_DIM + 2) * (BLOCK_DIM + 1) + shared_index_x] = pic_device_in[index + blockDim.x]; // load bottom edge
	}

	// Load corner pixels
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		pic_shared_step1[0] = pic_device_in[index - blockDim.x - 1]; // top-left corner
	}
	else if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
	{
		pic_shared_step1[BLOCK_DIM + 1] = pic_device_in[index - blockDim.x + 1]; // top-right corner
	}
	else if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
	{
		pic_shared_step1[(BLOCK_DIM + 2) * (BLOCK_DIM + 1)] = pic_device_in[index + blockDim.x - 1]; // bottom-left corner
	}
	else if (threadIdx.x == blockDim.x -1 && threadIdx.y == blockDim.y - 1)
	{
		pic_shared_step1[(BLOCK_DIM + 2) * (BLOCK_DIM + 1) + BLOCK_DIM + 1] = pic_device_in[index + blockDim.x + 1]; // bottom-right corner
	}

	__syncthreads(); // snychronize threads after loading shared memory

	// Apply anti-aliasing K times
	for (int iter = 0; iter < k; iter++)
	{
		// calculate the new value of the current pixel based on the average of its neighbors
		float sum = 0.0f;
		for (int i = -1; i <= 1; i++)
		{
			for (int j = -1; j <= 1; j++)
			{
				sum += pic_shared_step1[(shared_index_y + i) * (BLOCK_DIM + 2) + shared_index_x + j];
			}
		}

		float avg = sum / 9.0f; // average of 9 neighboring pixels

		// update the output image
		pic_shared_step2[shared_index_y * (BLOCK_DIM + 2) + shared_index_x] = avg;

		__syncthreads(); // sync threads after updating shared memory

		// fix the swap

		// swap shared memory arrays for the next iteration
		float* temp = pic_shared_step1;
		//pic_shared_step1 = pic_shared_step2;
		//pic_shared_step2 = temp;

	}

	// write the final result to global memory
	pic_device_out[index] = pic_shared_step1[shared_index_y * (BLOCK_DIM + 2) + shared_index_x];
}

static int AntiAliasing(float* pic_host_in, float* pic_host_out, int k)
{
	float* pic_device_in = nullptr;
	float* pic_device_out = nullptr;
	int size;
	dim3 block_count;
	dim3 block_size;

	size = static_cast<unsigned long long>(PICTURE_DIM) * PICTURE_DIM * sizeof(float);

	block_count.x = (int)ceil((float)PICTURE_DIM / (float)BLOCK_DIM);
	block_count.y = block_count.x;

	block_size.x = BLOCK_DIM;
	block_size.y = BLOCK_DIM;


	hipMalloc((void**)&pic_device_in, size);
	hipMalloc((void**)&pic_device_out, size);

	hipMemcpy(pic_device_in, pic_host_in, size, hipMemcpyHostToDevice);

	// kernel
	AntiAliasingKernel<<<block_count, block_size>>>(pic_device_in, pic_device_out, k);

	hipMemcpy(pic_host_out, pic_device_out, size, hipMemcpyDeviceToHost);

	hipFree(pic_device_in);
	hipFree(pic_device_out);

	hipDeviceReset();

	return 0;
}

static int FillPictures(float* pic_a, float* pic_b)
{
	// random slovo F

	for (int i = 0; i < PICTURE_DIM; i++)
	{
		for (int j = 0; j < PICTURE_DIM; j++)
		{
			pic_a[i * PICTURE_DIM + j] = 0.0;
			pic_b[i * PICTURE_DIM + j] = 0.0;
		}
	}

	for (int i = 9; i < 90; i++)
	{
		for (int j = 29; j < 40; j++)
		{
			pic_a[i * PICTURE_DIM + j] = 1.0;
		}
	}

	for (int i = 9; i < 30; i++)
	{
		for (int j = 40; j < 60; j++)
		{
			pic_a[i * PICTURE_DIM + j] = 1.0;
		}
	}

	for (int i = 50; i < 70; i++)
	{
		for (int j = 40; j < 50; j++)
		{
			pic_a[i * PICTURE_DIM + j] = 1.0;
		}
	}

	pic_a[PICTURE_DIM * PICTURE_DIM - 3] = 7;
	pic_a[PICTURE_DIM * PICTURE_DIM - 4] = 7;
	pic_a[PICTURE_DIM * PICTURE_DIM - 5] = 7;
	pic_a[PICTURE_DIM * PICTURE_DIM - PICTURE_DIM - 3] = 7;

	return 0;
}

static int PrintOutSample(float* pic, int x_start, int y_start, int x_size, int y_size)
{
	if (x_start < 0 || y_start < 0 || x_size > PICTURE_DIM || y_size > PICTURE_DIM)
	{
		std::cout << "Error printing sample!\n";
		return 1;
	}

	for (int i = x_start; i < x_start + x_size; i++)
	{
		for (int j = y_start; j < y_start + y_size; j++)
		{
			std::cout << pic[i * PICTURE_DIM + j];
		}
		std::cout << std::endl;
	}

	return 0;
}

int april2023_zadatak1()
{
	float picture_a[PICTURE_DIM * PICTURE_DIM];
	float picture_b[PICTURE_DIM * PICTURE_DIM];

	FillPictures(picture_a, picture_b);

	AntiAliasing(picture_a, picture_b, 3);

	//TestResults(picture_a, picture_b);
	PrintOutSample(picture_a, 0, 0, PICTURE_DIM, PICTURE_DIM);

	std::cout << "------------------------" << std::endl;

	PrintOutSample(picture_b, 0, 0, PICTURE_DIM, PICTURE_DIM);

	return 0;
}

#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "jun2023zadatak2.cuh"

// Korišćenjem CUDA tehnologije, u programskom jeziku C/C++ napisati program koji nalazi minimalni element na glavnoj
// dijagonali kvadratne matrice A dimenzija n x n. Obratiti pažnju na efikasnost paralelizacije. Omogućiti pozivanje
// kernela za matrice proizvoljne veličine.

#define N 1940 // max 12228 zbog deljene memorije
#define BLK_SIZE 1024

__global__ void MinimalElementKernel(float* device_matrix, float* device_min_element, int* device_array_size)
{
	__shared__ float main_diagonal[BLK_SIZE];
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

	main_diagonal[threadIdx.x] = (index < N)? A_device[index * N + index] : FLT_MAX;

	__syncthreads();




}

static int MinimalElement(float* host_matrix, float* host_min_element)
{
	int size = N * N * sizeof(float);
	const int block_count = (int)ceil((float)N / (float)BLK_SIZE);

	float* device_matrix = nullptr;
	float* device_min_element = nullptr;
	float* device_array_size = nullptr;

	

	hipMalloc((void**)&device_matrix, size);
	hipMalloc((void**)&device_min_element, sizeof(float));
	hipMalloc((void**)&device_array_size, sizeof(float));
	
	hipMemcpy(device_matrix, host_matrix, size, hipMemcpyHostToDevice);


	MinimalElementKernel<<<block_count, N>>>(A_device, min_element_device);

	hipMemcpy(host_min_element, device_min_element, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(A_device);
	hipFree(min_element_device);

	hipDeviceReset();

	return 0;
}

static int FillMatrix(float* A)
{
	constexpr float min = std::numeric_limits<float>::min();
	constexpr float max = std::numeric_limits<float>::max();
	constexpr float range = max - min;

	srand((unsigned int)time(0));

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[i * N + j] = min + static_cast<float>(rand()) / (RAND_MAX / range);
		}
	}

	return 0;
}

static bool TestResult(float* A, float* min_element_returned)
{
	for (auto i = 0; i < N; i++)
	{
		if (A[i * N + i] < *min_element_returned)
		{
			return false;
		}
	}

	return true;
}

int jun2023zadatak2()
{
	float A[N * N];
	float min_element_host;

	FillMatrix(A);

	MinimalElement(A, &min_element_host);

	TestResult(A, &min_element_host);

	return 0;
}